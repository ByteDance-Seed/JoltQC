#include "hip/hip_runtime.h"
/*
# Copyright 2025 ByteDance Inc.
#
# Licensed under the Apache License, Version 2.0 (the "License");
# you may not use this file except in compliance with the License.
# You may obtain a copy of the License at
#
#     http://www.apache.org/licenses/LICENSE-2.0
#
# Unless required by applicable law or agreed to in writing, software
# distributed under the License is distributed on an "AS IS" BASIS,
# WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
# See the License for the specific language governing permissions and
# limitations under the License.
#
*/

// Portions of this file adapted from GPU4PySCF v1.4 (https://github.com/pyscf/gpu4pyscf)
// Copyright 2025 PySCF developer.
// Licensed under the Apache License, Version 2.0.

// Ensure 64-bit integer width across platforms
typedef unsigned long long uint64_t;
constexpr float minval = -36.8; // exp(-36.8) ~ 1e-16

__forceinline__ __device__
int global_offset(int* batch_head, int val){
    // Calculate the cumulative sum of the count array
    constexpr int warp_size = 32;
    constexpr int num_warps = threads / warp_size;

    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int lane  = tid & (warp_size - 1);    
    const int warp  = tid / warp_size;
    int inclusive = val;
    for (int ofs = 1; ofs < warp_size; ofs <<= 1) {
        int n = __shfl_up_sync(0xffffffff, inclusive, ofs);
        if (lane >= ofs) inclusive += n;                
    }

    __shared__ int warp_tot[num_warps];  
    if (lane == warp_size - 1) warp_tot[warp] = inclusive;  
    __syncthreads(); 

    if (warp == 0) {
        int wval = (lane < num_warps) ? warp_tot[lane] : 0;
#pragma unroll
        for (int ofs = 1; ofs < warp_size; ofs <<= 1) {
            int n = __shfl_up_sync(0xffffffff, wval, ofs);
            if (lane >= ofs) wval += n;
        }
        if (lane < num_warps) warp_tot[lane] = wval;
    }
    __syncthreads();

    // Block-exclusive prefix for this thread
    const int warp_offset      = (warp == 0) ? 0 : warp_tot[warp - 1];
    const int inclusive_block  = warp_offset + inclusive;
    const int exclusive_block  = inclusive_block - val;

    // --- block total is the last warp's inclusive sum
    const int block_total = warp_tot[num_warps - 1];

    if (block_total == 0) return 0;

    // Single atomic to reserve a global range
    __shared__ int base;
    if (tid == 0) base = atomicAdd(batch_head, block_total);
    __syncthreads();

    return base + exclusive_block;
}


extern "C" __global__ 
void screen_jk_tasks(ushort4 *shl_quartet_idx, int *batch_head, const int nbas, 
    const int * __restrict__ tile_ij_mapping, 
    const int * __restrict__ tile_kl_mapping, 
    const int ntiles_ij1, const int ntiles_kl1,
    const float * __restrict__ q_cond,
    const float * __restrict__ dm_cond, 
    const float cutoff, const float cutoff_fp64)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    int ij = blockIdx.x * blockDim.x + tx;
    int kl = blockIdx.y * blockDim.y + ty;

    bool active = true;
    if (ij >= ntiles_ij1 || kl >= ntiles_kl1){
        ij = 0;
        kl = 0;
        active = false;
    }

    // Load tile mappings only if active to avoid OOB when mappings are empty
    const int nbas_tiles = nbas / TILE;
    int tile_ij = 0, tile_kl = 0;
    int tile_i = 0, tile_j = 0, tile_k = 0, tile_l = 0;
    int ish0 = 0, jsh0 = 0, ksh0 = 0, lsh0 = 0;
    if (active) {
        tile_ij = tile_ij_mapping[ij];
        tile_kl = tile_kl_mapping[kl];
        // Optimize division and modulo operations
        tile_i = tile_ij / nbas_tiles;
        tile_j = tile_ij - tile_i * nbas_tiles;  // Replace modulo with subtraction
        tile_k = tile_kl / nbas_tiles;
        tile_l = tile_kl - tile_k * nbas_tiles;  // Replace modulo with subtraction
        
        ish0 = tile_i * TILE;
        jsh0 = tile_j * TILE;
        ksh0 = tile_k * TILE;
        lsh0 = tile_l * TILE;
    }
    // Removed unused ish1/jsh1/ksh1/lsh1 to reduce register pressure
    constexpr int align = 2;
    constexpr int align2 = align*align;

    // Fragment-level screening: store fragment indices, not individual items
    constexpr int frag_size = (TILE / align);
    constexpr int num_frags = frag_size * frag_size * frag_size * frag_size;
    constexpr int frag_mask_size = (num_frags + 63) / 64;
    uint64_t frag_mask_fp32[frag_mask_size] = {0};
    uint64_t frag_mask_fp64[frag_mask_size] = {0};

    int count_fp32 = 0;
    int count_fp64 = 0;
    if (active){
        for (int i0 = 0; i0 < TILE; i0 += align){
        for (int j0 = 0; j0 < TILE; j0 += align){
        for (int k0 = 0; k0 < TILE; k0 += align){
        for (int l0 = 0; l0 < TILE; l0 += align){
            const int bas_ij = (ish0 + i0) * nbas + (jsh0 + j0);
            const int bas_kl = (ksh0 + k0) * nbas + (lsh0 + l0);
            if (bas_ij < bas_kl) continue;
            if (jsh0 + j0 > ish0 + i0) continue;
            if (ksh0 + k0 > ish0 + i0) continue;
            if (lsh0 + l0 > ksh0 + k0) continue;
            float dm_kl[align2];
            float dm_ij[align2];
            float dm_jl[align2];
            float dm_ik[align2];
            float dm_jk[align2];
            float dm_il[align2];

            float q_kl[align2];
            float q_ij[align2];
            for (int ii = 0; ii < align; ++ii){
                const int ish = ish0 + i0 + ii;
                for (int jj = 0; jj < align; ++jj){
                    const int jsh = jsh0 + j0 + jj;
                    bool mask = ish < nbas && jsh < nbas;
                    float dm = mask ? dm_cond[ish * nbas + jsh] : minval;
                    float q = mask ? q_cond[ish * nbas + jsh] : minval;
                    dm_ij[ii*align + jj] = dm;
                    q_ij[ii*align + jj] = q;
                }
                if constexpr(do_k){
                    for (int kk = 0; kk < align; ++kk){
                        const int ksh = ksh0 + k0 + kk;
                        bool mask = ish < nbas && ksh < nbas;
                        float dm = mask ? dm_cond[ish * nbas + ksh] : minval;
                        dm_ik[ii*align + kk] = dm;
                    }
                    for (int ll = 0; ll < align; ++ll){
                        const int lsh = lsh0 + l0 + ll;
                        bool mask = ish < nbas && lsh < nbas;
                        float dm = mask ? dm_cond[ish * nbas + lsh] : minval;
                        dm_il[ii*align + ll] = dm;
                    }
                }
            }
            if constexpr(do_k){
                for (int jj = 0; jj < align; ++jj){
                    const int jsh = jsh0 + j0 + jj;
                    for (int kk = 0; kk < align; ++kk){
                        const int ksh = ksh0 + k0 + kk;
                        bool mask = jsh < nbas && ksh < nbas;
                        float dm = mask ? dm_cond[jsh * nbas + ksh] : minval;
                        dm_jk[jj*align + kk] = dm;
                    }

                    for (int ll = 0; ll < align; ++ll){
                        const int lsh = lsh0 + l0 + ll;
                        bool mask = jsh < nbas && lsh < nbas;
                        float dm = mask ? dm_cond[jsh * nbas + lsh] : minval;
                        dm_jl[jj*align + ll] = dm;
                    }
                }
            }

            for (int kk = 0; kk < align; ++kk){
                const int ksh = ksh0 + k0 + kk;
                for (int ll = 0; ll < align; ++ll){
                    const int lsh = lsh0 + l0 + ll;
                    const int kl = kk*align + ll;
                    const int kl_sh = ksh * nbas + lsh;
                    bool mask = ksh < nbas && lsh < nbas;
                    float dm = mask ? dm_cond[ksh * nbas + lsh] : minval;
                    float q = mask ? q_cond[ksh * nbas + lsh] : minval;
                    dm_kl[kl] = dm;
                    q_kl[kl] = q;
                }
            }
            
            bool select_fp32 = false;
            bool select_fp64 = false;
            for (int ii = 0; ii < align; ++ii){
            for (int jj = 0; jj < align; ++jj){
            for (int kk = 0; kk < align; ++kk){
            for (int ll = 0; ll < align; ++ll){
                const float q_ijkl = q_ij[ii*align + jj] + q_kl[kk*align + ll];
                float d_large = minval;

                if constexpr(do_k){
                    const float d_ik = dm_ik[ii*align + kk];
                    const float d_jk = dm_jk[jj*align + kk];
                    const float d_il = dm_il[ii*align + ll];
                    const float d_jl = dm_jl[jj*align + ll];
                    d_large = max(d_large, d_ik);
                    d_large = max(d_large, d_jk);
                    d_large = max(d_large, d_il);
                    d_large = max(d_large, d_jl);
                }
                if constexpr(do_j){
                    const float d_ij = dm_ij[ii*align + jj];
                    const float d_kl = dm_kl[kk*align + ll];
                    d_large = max(d_large, d_ij);
                    d_large = max(d_large, d_kl);
                }

                const float dq = q_ijkl + d_large;
                select_fp32 |= (dq > cutoff) && (dq <= cutoff_fp64);
                select_fp64 |= (dq > cutoff_fp64);
            }}}}

            if (!(select_fp32 || select_fp64)) {
                // Skip entire fragment if no combinations are selected
                continue;
            }

            // Mark the fragment - FP64 takes priority over FP32
            // If any item in fragment needs FP64, entire fragment goes to FP64
            const int frag_i = i0 / align;
            const int frag_j = j0 / align;
            const int frag_k = k0 / align;
            const int frag_l = l0 / align;
            const uint64_t frag_idx = ((frag_i * frag_size + frag_j) * frag_size + frag_k) * frag_size + frag_l;
            const uint64_t frag_word = frag_idx >> 6;
            const uint64_t frag_bit = frag_idx & 63;
            const uint64_t frag_bitmask = 1ull << frag_bit;

            if (select_fp64) {
                // Fragment needs FP64 precision
                frag_mask_fp64[frag_word] |= frag_bitmask;
                count_fp64 += align2*align2;
            } else {
                // Fragment only needs FP32 precision
                frag_mask_fp32[frag_word] |= frag_bitmask;
                count_fp32 += align2*align2;
            }
        }}}}
    }

    // Check if entire block has no work - all threads must participate
    __shared__ bool has_work;
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        has_work = false;
    }
    __syncthreads();

    if (count_fp32 > 0 || count_fp64 > 0) {
        atomicOr((int*)&has_work, 1);
    }
    __syncthreads();

    if (!has_work) {
        return;
    }

    int offset_fp32 = global_offset(batch_head+1, count_fp32);
    int offset_fp64 = global_offset(batch_head+2, -count_fp64) - 1;
    
    if (active){
#pragma unroll
        for (int frag_i = 0; frag_i < frag_size; frag_i++){
        for (int frag_j = 0; frag_j < frag_size; frag_j++){
        for (int frag_k = 0; frag_k < frag_size; frag_k++){
        for (int frag_l = 0; frag_l < frag_size; frag_l++){
            const uint64_t frag_idx = ((frag_i * frag_size + frag_j) * frag_size + frag_k) * frag_size + frag_l;
            const uint64_t frag_word = frag_idx >> 6;
            const uint64_t frag_bit = frag_idx & 63;

            const bool sel_fp32 = (frag_mask_fp32[frag_word] >> frag_bit) & 1ull;
            const bool sel_fp64 = (frag_mask_fp64[frag_word] >> frag_bit) & 1ull;

            if (sel_fp32 || sel_fp64) {
                // Output all items in this fragment
                for (int ii = 0; ii < align; ii++){
                for (int jj = 0; jj < align; jj++){
                for (int kk = 0; kk < align; kk++){
                for (int ll = 0; ll < align; ll++){
                    const int i = frag_i * align + ii;
                    const int j = frag_j * align + jj;
                    const int k = frag_k * align + kk;
                    const int l = frag_l * align + ll;

                    ushort4 sq;
                    sq.x = ish0 + i;
                    sq.y = jsh0 + j;
                    sq.z = ksh0 + k;
                    sq.w = lsh0 + l;

                    if (sel_fp64) {
                        shl_quartet_idx[offset_fp64] = sq;
                        --offset_fp64;
                    } else if (sel_fp32) {
                        shl_quartet_idx[offset_fp32] = sq;
                        ++offset_fp32;
                    }
                }}}}
            }
        }}}}
    }
}
