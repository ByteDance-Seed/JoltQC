#include "hip/hip_runtime.h"
/*
# Copyright 2025 ByteDance Inc.
#
# Licensed under the Apache License, Version 2.0 (the "License");
# you may not use this file except in compliance with the License.
# You may obtain a copy of the License at
#
#     http://www.apache.org/licenses/LICENSE-2.0
#
# Unless required by applicable law or agreed to in writing, software
# distributed under the License is distributed on an "AS IS" BASIS,
# WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
# See the License for the specific language governing permissions and
# limitations under the License.
#
*/

// Portions of this file adapted from GPU4PySCF v1.4 (https://github.com/pyscf/gpu4pyscf)
// Copyright 2025 PySCF developer.
// Licensed under the Apache License, Version 2.0.

constexpr int DEGREE = 13;
constexpr int DEGREE1 = (DEGREE+1);
constexpr int INTERVALS = 40;

constexpr DataType SQRTPIE4 = .8862269254527580136;
constexpr DataType PIE4     = .7853981633974483096;

__device__ __forceinline__
static void rys_roots(DataType x, DataType *rw, int rt_id, const int stride, DataType theta, DataType omega)
{
    constexpr DataType one = 1.0;
    constexpr DataType two = 2.0;
    constexpr DataType half= .5;
    const int stride2 = stride * 2;

    x *= theta;

    const DataType omega2 = omega*omega;
    const DataType theta_fac = omega2 / (omega2 + theta);
    const DataType sqrt_theta_fac = sqrt(theta_fac);

    if constexpr (rys_type > 0){
        x *= theta_fac;
    }
    
    if (x < 3.e-7){
#pragma unroll
        for (int i = rt_id; i < nroots; i += nthreads_per_sq)  {
            DataType root = ROOT_SMALLX_R0[i] + ROOT_SMALLX_R1[i] * x;
            DataType weight = ROOT_SMALLX_W0[i] + ROOT_SMALLX_W1[i] * x;
            if constexpr(rys_type > 0){
                root *= theta_fac;
                weight *= sqrt_theta_fac;
            }
            rw[i*stride2         ] = root;
            rw[i*stride2 + stride] = weight;
        }
        return;
    }
    
    if (x > 35+nroots*5) {
        const DataType inv_x = one / x; 
        const DataType t = sqrt(PIE4 * inv_x);
#pragma unroll
        for (int i = rt_id; i < nroots; i += nthreads_per_sq)  {
            DataType root = ROOT_LARGEX_R_DATA[i] * inv_x;
            DataType weight = ROOT_LARGEX_W_DATA[i] * t;
            if constexpr(rys_type > 0){
                root *= theta_fac;
                weight *= sqrt_theta_fac;
            }
            rw[i*stride2         ] = root;
            rw[i*stride2 + stride] = weight;
        }
        return;
    }

    if constexpr(nroots == 1) {
        const DataType tt = sqrt(x);                       // 1 sqrt
        const DataType erf_tt = erf(tt);                   // 1 erf
        const DataType e = exp(-x);                        // 1 exp

        const DataType inv_tt = SQRTPIE4 / tt;             // 1 div
        const DataType fmt0 = inv_tt * erf_tt;

        DataType weight = fmt0;
        const DataType fmt1 = (half / x) * (fmt0 - e);     // 1 div
        DataType root = fmt1 / fmt0;                               // 1 div
        
        if constexpr(rys_type > 0){
            root *= theta_fac;
            weight *= sqrt_theta_fac;
        }
        rw[0] = root;
        rw[stride] = weight;
        return;
    }

    const int it = (int)(x * .4f);
    const DataType u = (x - it * DataType(2.5)) * DataType(0.8) - DataType(1.);
    const DataType u2 = u * two;
    const DataType *datax = ROOT_RW_DATA + it;
#pragma unroll
    for (int i = rt_id; i < nroots; i += nthreads_per_sq) {
        {
            const DataType *c = datax + (2*i) * DEGREE1 * INTERVALS;
            DataType c0 = c[DEGREE*INTERVALS];
            DataType c1 = c[DEGREE*INTERVALS - INTERVALS];
    #pragma unroll
            for (int n = DEGREE-2; n > 0; n-=2) {
                const DataType c2 = c[n*INTERVALS] - c1;
                const DataType c3 = c0 + c1*u2;
                c1 = c2 + c3*u2;
                c0 = c[n*INTERVALS - INTERVALS] - c3;
            }
            DataType root = c0 + c1*u;
            if constexpr(rys_type > 0){
                root *= theta_fac;
            }
            rw[i*stride2] = root;
        }

        {
            const DataType *c = datax + (2*i+1) * DEGREE1 * INTERVALS;
            DataType c0 = c[DEGREE*INTERVALS];
            DataType c1 = c[DEGREE*INTERVALS - INTERVALS];
    #pragma unroll
            for (int n = DEGREE-2; n > 0; n-=2) {
                const DataType c2 = c[n*INTERVALS] - c1;
                const DataType c3 = c0 + c1*u2;
                c1 = c2 + c3*u2;
                c0 = c[n*INTERVALS - INTERVALS] - c3;
            }
            DataType weight = c0 + c1*u;
            if constexpr(rys_type > 0){
                weight *= sqrt_theta_fac;
            }
            rw[i*stride2 + stride] = weight;
        }
    }
}
