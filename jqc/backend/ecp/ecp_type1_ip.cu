#include "hip/hip_runtime.h"
/*
# Copyright 2025 ByteDance Inc.
#
# Licensed under the Apache License, Version 2.0 (the "License");
# you may not use this file except in compliance with the License.
# You may obtain a copy of the License at
#
#     http://www.apache.org/licenses/LICENSE-2.0
#
# Unless required by applicable law or agreed to in writing, software
# distributed under the License is distributed on an "AS IS" BASIS,
# WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
# See the License for the specific language governing permissions and
# limitations under the License.
#
*/

// Portions of this file adapted from GPU4PySCF (https://github.com/pyscf/gpu4pyscf)
// Copyright 2021-2024 PySCF developer.
// Licensed under the Apache License, Version 2.0.

/*
 * OPTIMIZATION NOTE: Shared memory allocations in this file have been optimized
 * for better reuse across different kernel configurations by basing sizes on
 * global variables LI+2 and LJ+1 (injected from Python) instead of exact
 * template parameters LIT, LJT. This allows the same shared memory regions to
 * be reused for multiple angular momentum combinations, reducing memory
 * allocation overhead and improving kernel performance.
 */




extern "C" __global__
void type1_cart_ip1(double* __restrict__ gctr,
                const int* __restrict__ ao_loc, const int nao,
                const int* __restrict__ tasks, const int ntasks,
                const int* __restrict__ ecpbas, const int* __restrict__ ecploc,
                const DataType4* __restrict__ coords,
                const DataType2* __restrict__ coeff_exp,
                const int* __restrict__ atm, const double* __restrict__ env,
                const int npi, const int npj)
{
    const int task_id = blockIdx.x;
    if (task_id >= ntasks){
        return;
    }

    const int ish = tasks[task_id];
    const int jsh = tasks[task_id + ntasks];
    const int ksh = tasks[task_id + 2*ntasks];
    const int ioff = ao_loc[ish];
    const int joff = ao_loc[jsh];
    const int ecp_id = ecpbas[ECP_ATOM_ID+ecploc[ksh]*BAS_SLOTS];
    gctr += ioff*nao + joff + 3*ecp_id*nao*nao;
    
    constexpr int nfi = (LI+1) * (LI+2) / 2;
    constexpr int nfj = (LJ+1) * (LJ+2) / 2;
    extern __shared__ char shared_mem[];

    // Allocate gctr_smem from shared memory
    double* gctr_smem = reinterpret_cast<double*>(shared_mem);
    size_t gctr_offset = nfi * nfj * 3 * sizeof(double);

    for (int ij = threadIdx.x; ij < nfi*nfj*3; ij+=blockDim.x){
        gctr_smem[ij] = 0.0;
    }
    __syncthreads();

    constexpr int nfi_max = (LI+2)*(LI+3)/2;
    constexpr int nfj_max = (LJ+1)*(LJ+2)/2;

    // Allocate buffer and kernel shared memory
    double* buf = reinterpret_cast<double*>(shared_mem + gctr_offset);
    char* kernel_shared_mem = shared_mem + gctr_offset + 3 * nfi_max * nfj_max * sizeof(double);

    // Accumulate derivative contributions with respect to AO i.
    // j-side contributions are accumulated via (j,i) tasks in host tasking (full tasks).
    // Use LI+1 for orderi=1 to match unrolled cache pattern
    type1_cart_kernel<LI+1, LJ, 1, 0>(buf, ish, jsh, ksh, ecpbas, ecploc, 
        coords, coeff_exp, atm, env, npi, npj, kernel_shared_mem);
    __syncthreads();
    _li_down<LI, LJ>(gctr_smem, buf);
    __syncthreads();

    if constexpr (LI > 0){
        // Use LI-1 for orderi=0 companion
        set_shared_memory(buf, 3 * nfi_max * nfj_max);
        type1_cart_kernel<LI-1, LJ, 0, 0>(buf, ish, jsh, ksh, ecpbas, ecploc, 
            coords, coeff_exp, atm, env, npi, npj, kernel_shared_mem);
        __syncthreads();
        _li_up<LI, LJ>(gctr_smem, buf);
        __syncthreads();
    }

    for (int ij = threadIdx.x; ij < nfi*nfj; ij+=blockDim.x){
        const int i = ij%nfi;
        const int j = ij/nfi;
        double *gx = gctr;
        double *gy = gctr +   nao*nao;
        double *gz = gctr + 2*nao*nao;
        atomicAdd(gx+i*nao+j, gctr_smem[ij]);
        atomicAdd(gy+i*nao+j, gctr_smem[ij+nfi*nfj]);
        atomicAdd(gz+i*nao+j, gctr_smem[ij+2*nfi*nfj]);
    }
    return;
}

