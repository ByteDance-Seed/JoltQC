
#include <hip/hip_runtime.h>
/*
# Copyright 2025 ByteDance Inc.
#
# Licensed under the Apache License, Version 2.0 (the "License");
# you may not use this file except in compliance with the License.
# You may obtain a copy of the License at
#
#     http://www.apache.org/licenses/LICENSE-2.0
#
# Unless required by applicable law or agreed to in writing, software
# distributed under the License is distributed on an "AS IS" BASIS,
# WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
# See the License for the specific language governing permissions and
# limitations under the License.
#
*/

// # Portions of this file adapted from GPU4PySCF (https://github.com/pyscf/gpu4pyscf)
// # Copyright 2025 PySCF developer.
// # Licensed under the Apache License, Version 2.0.

using DataType = float;

#define NG_PER_BLOCK      128

extern "C" __global__
void vv10_kernel(double *Fvec, double *Uvec, double *Wvec,
    const double *vvcoords, const double *coords,
    const double *W0p, const double *W0, const double *K,
    const double *Kp, const double *RpW,
    const int vvngrids, const int ngrids)
{
    // grid id
    int grid_id = blockIdx.x * blockDim.x + threadIdx.x;
    const bool active = grid_id < ngrids;
    DataType xi, yi, zi;
    DataType W0i, Ki;
    if (active){
        xi = coords[grid_id];
        yi = coords[ngrids + grid_id];
        zi = coords[2*ngrids + grid_id];
        W0i = W0[grid_id];
        Ki = K[grid_id];
    }

    double F = 0.0;
    double U = 0.0;
    double W = 0.0;

    const double *xj = vvcoords;
    const double *yj = vvcoords + vvngrids;
    const double *zj = vvcoords + 2*vvngrids;

    __shared__ DataType xj_smem[NG_PER_BLOCK];
    __shared__ DataType yj_smem[NG_PER_BLOCK];
    __shared__ DataType zj_smem[NG_PER_BLOCK];
    __shared__ DataType Kp_smem[NG_PER_BLOCK];
    __shared__ DataType W0p_smem[NG_PER_BLOCK];
    __shared__ DataType RpW_smem[NG_PER_BLOCK];

    const int tx = threadIdx.x;

    for (int j = 0; j < vvngrids; j+=blockDim.x) {
        int idx = j + tx;
        if (idx < vvngrids){
            xj_smem[tx] = xj[idx];
            yj_smem[tx] = yj[idx];
            zj_smem[tx] = zj[idx];
            Kp_smem[tx] = Kp[idx];
            W0p_smem[tx] = W0p[idx];
            RpW_smem[tx] = RpW[idx];
        }
        __syncthreads();

        for (int l = 0, M = min(NG_PER_BLOCK, vvngrids - j); l < M; ++l){
            DataType DX = xj_smem[l] - xi;
            DataType DY = yj_smem[l] - yi;
            DataType DZ = zj_smem[l] - zi;
            DataType R2 = DX*DX + DY*DY + DZ*DZ;

            DataType gp = R2 * W0p_smem[l] + Kp_smem[l];
            DataType g  = R2*W0i + Ki;
            DataType gt = g + gp;
            DataType ggt = g*gt;
            DataType g_gt = g + gt;
            DataType T = RpW_smem[l] / (gp*ggt*ggt);

            F += T * ggt;
            U += T * g_gt;
            W += T * R2 * g_gt;
        }
        __syncthreads();
    }
    if(active){
        Fvec[grid_id] = F * -1.5;
        Uvec[grid_id] = U;
        Wvec[grid_id] = W;
    }

}