#include "hip/hip_runtime.h"
/*
# Copyright 2025 ByteDance Inc.
#
# Licensed under the Apache License, Version 2.0 (the "License");
# you may not use this file except in compliance with the License.
# You may obtain a copy of the License at
#
#     http://www.apache.org/licenses/LICENSE-2.0
#
# Unless required by applicable law or agreed to in writing, software
# distributed under the License is distributed on an "AS IS" BASIS,
# WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
# See the License for the specific language governing permissions and
# limitations under the License.
#
*/

// # Portions of this file adapted from GPU4PySCF (https://github.com/pyscf/gpu4pyscf)
// # Copyright 2025 PySCF developer.
// # Licensed under the Apache License, Version 2.0.

// DataType will be defined by the calling code

// NG_PER_BLOCK will be defined by the calling code

extern "C" __global__
void vv10_kernel(double *Fvec, double *Uvec, double *Wvec,
    const double * __restrict__ vvcoords, const double * __restrict__ coords,
    const double * __restrict__ W0p, const double * __restrict__ W0, 
    const double *__restrict__ K, const double *__restrict__ Kp, 
    const double *__restrict__ RpW, const int vvngrids, const int ngrids)
{
    // grid id - assume 256-aligned grids (guaranteed by padding)
    const int grid_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load grid data (no bounds check needed due to 256-alignment guarantee)
    const DataType xi = coords[grid_id];
    const DataType yi = coords[ngrids + grid_id];
    const DataType zi = coords[2*ngrids + grid_id];
    const DataType W0i = W0[grid_id];
    const DataType Ki = K[grid_id];

    double F = 0.0;
    double U = 0.0;
    double W = 0.0;

    const double *xj = vvcoords;
    const double *yj = vvcoords + vvngrids;
    const double *zj = vvcoords + 2*vvngrids;

    __shared__ DataType xj_smem[NG_PER_BLOCK];
    __shared__ DataType yj_smem[NG_PER_BLOCK];
    __shared__ DataType zj_smem[NG_PER_BLOCK];
    __shared__ DataType Kp_smem[NG_PER_BLOCK];
    __shared__ DataType W0p_smem[NG_PER_BLOCK];
    __shared__ DataType RpW_smem[NG_PER_BLOCK];

    const int tx = threadIdx.x;

    for (int j = 0; j < vvngrids; j+=blockDim.x) {
        int idx = j + tx;
        
        // Load data directly (no bounds check needed due to 256-alignment guarantee)
        xj_smem[tx] = xj[idx];
        yj_smem[tx] = yj[idx];
        zj_smem[tx] = zj[idx];
        Kp_smem[tx] = Kp[idx];
        W0p_smem[tx] = W0p[idx];
        RpW_smem[tx] = RpW[idx];
        
        __syncthreads();

        // Compute VV10 interaction
#pragma unroll 16
        for (int l = 0; l < NG_PER_BLOCK; ++l){
            const DataType DX = xj_smem[l] - xi;
            const DataType DY = yj_smem[l] - yi;
            const DataType DZ = zj_smem[l] - zi;
            const DataType R2 = DX*DX + DY*DY + DZ*DZ;

            const DataType gp = R2 * W0p_smem[l] + Kp_smem[l];
            const DataType g  = R2*W0i + Ki;
            const DataType gt = g + gp;
            const DataType ggt = g*gt;
            const DataType g_gt = g + gt;
            
            // Add safety check for division by zero
            const DataType denominator = gp*ggt*ggt;
            if (denominator > 1e-20) {
                const DataType T = RpW_smem[l] / denominator;
                F += T * ggt;
                U += T * g_gt;
                W += T * R2 * g_gt;
            }
        }
        __syncthreads();
    }
    
    // Store results (no bounds check needed due to 256-alignment guarantee)
    Fvec[grid_id] = F * -1.5;
    Uvec[grid_id] = U;
    Wvec[grid_id] = W;

}