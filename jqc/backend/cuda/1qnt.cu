#include "hip/hip_runtime.h"
/*
# Copyright 2025 ByteDance Inc.
#
# Licensed under the Apache License, Version 2.0 (the "License");
# you may not use this file except in compliance with the License.
# You may obtain a copy of the License at
#
#     http://www.apache.org/licenses/LICENSE-2.0
#
# Unless required by applicable law or agreed to in writing, software
# distributed under the License is distributed on an "AS IS" BASIS,
# WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
# See the License for the specific language governing permissions and
# limitations under the License.
#
*/

// Portions of this file adapted from GPU4PySCF v1.4 (https://github.com/pyscf/gpu4pyscf)
// Copyright 2025 PySCF developer.
// Licensed under the Apache License, Version 2.0.

// 2*pi**2.5
constexpr DataType PI_FAC = 34.98683665524972497;
constexpr DataType half = .5;
constexpr DataType one = 1.0;
constexpr DataType zero = 0.0;
constexpr int nprim_max = 16;

struct __align__(4*sizeof(DataType)) DataType4 {
    DataType x, y, z, w;
};

struct __align__(2*sizeof(DataType)) DataType2 {
    DataType c, e;
};

extern "C" __global__
void rys_jk(const int nbas,  
        const int* __restrict__ ao_loc, 
        const DataType4* __restrict__ coords,
        const DataType2* __restrict__ coeff_exp,
        DataType* dm, 
        double* vj, 
        double* vk,
        const DataType omega,
        const ushort4* __restrict__ shl_quartet_idx, 
        const int ntasks)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int task_id = blockIdx.x * blockDim.x + threadIdx.x;

    constexpr int nfi = (li+1)*(li+2)/2;
    constexpr int nfj = (lj+1)*(lj+2)/2;
    constexpr int nfk = (lk+1)*(lk+2)/2;
    constexpr int nfl = (ll+1)*(ll+2)/2;

    constexpr int nfij = nfi*nfj;
    constexpr int nfkl = nfk*nfl;
    constexpr int nfik = nfi*nfk;
    constexpr int nfil = nfi*nfl;
    constexpr int nfjk = nfj*nfk;
    constexpr int nfjl = nfj*nfl;

    constexpr int nti = (nfi + fragi - 1) / fragi;
    constexpr int ntj = (nfj + fragj - 1) / fragj;
    constexpr int ntk = (nfk + fragk - 1) / fragk;
    constexpr int ntl = (nfl + fragl - 1) / fragl;
    constexpr int nt_active = nti * ntj * ntk * ntl;

    constexpr int tstride_l = 1;
    constexpr int tstride_k = fragl;
    constexpr int tstride_j = fragl * fragk;
    constexpr int tstride_i = fragl * fragk * fragj;
    constexpr int frag_size = fragi*fragj*fragk*fragl;

    const int t_i = (ty % nti);
    const int t_j = (ty / nti % ntj);
    const int t_k = (ty / (nti*ntj) % ntk);
    const int t_l = (ty / (nti*ntj*ntk));

    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    constexpr int gx_stride = smem_stride;
    constexpr int g_stride = 3 * gx_stride;

    // shape of g, (gsize, 3, nsq_per_block)
    constexpr int stride_i = g_stride;
    constexpr int stride_j = stride_i * (li+1);
    constexpr int stride_k = stride_j * (lj+1);
    constexpr int stride_l = stride_k * (lk+1);

    // Dynamic shared memory buffer
    extern __shared__ DataType shared_memory[];

    int kl_idx[fragk*fragl], kl_idy[fragk*fragl], kl_idz[fragk*fragl];
    for (int reg_k = 0; reg_k < fragk; reg_k++){
        const int k = t_k * fragk + reg_k;
        for (int reg_l = 0; reg_l < fragl; reg_l++){
            const int l = t_l * fragl + reg_l;
            if (ty < nt_active) {
                const int kl = reg_l + reg_k * fragl;
                const uint32_t addr = k_idx[k] + l_idx[l];
                const uint32_t kl_x =  addr        & 0x3FF;      // 10 low-order bits
                const uint32_t kl_y = (addr >> 10) & 0x3FF;      // next 10 bits
                const uint32_t kl_z = (addr >> 20) & 0x3FF;      // next 10 bits
                kl_idx[kl] = kl_x * g_stride;
                kl_idy[kl] = kl_y * g_stride + gx_stride;
                kl_idz[kl] = kl_z * g_stride + 2*gx_stride;
            }
        }
    }

    const bool active = (task_id < ntasks);
    ushort4 sq = {0,0,0,0};
    if (active) {
        sq = shl_quartet_idx[task_id];
    }
    
    const int ish = (int)sq.x;
    const int jsh = (int)sq.y;
    const int ksh = (int)sq.z;
    const int lsh = (int)sq.w;
    
    DataType fac_sym = active ? PI_FAC : zero;
    fac_sym *= (ish == jsh) ? half : one;
    fac_sym *= (ksh == lsh) ? half : one;
    fac_sym *= (ish*nbas+jsh == ksh*nbas+lsh) ? half : one;

    const DataType4 ri = coords[ish];
    const DataType4 rj = coords[jsh];
    const DataType4 rk = coords[ksh];
    const DataType4 rl = coords[lsh];

    const DataType rjri0 = rj.x - ri.x;
    const DataType rjri1 = rj.y - ri.y;
    const DataType rjri2 = rj.z - ri.z;
    const DataType rr_ij = rjri0*rjri0 + rjri1*rjri1 + rjri2*rjri2;
    const DataType rlrk0 = rl.x - rk.x;
    const DataType rlrk1 = rl.y - rk.y;
    const DataType rlrk2 = rl.z - rk.z;
    const DataType rr_kl = rlrk0*rlrk0 + rlrk1*rlrk1 + rlrk2*rlrk2;

    // Cache coefficients and precompute cicj values in shared memory for better performance
    DataType2 reg_cei[npi], reg_cej[npj];
    for (int ip = 0; ip < npi; ip++){
        const int ish_ip = ip + ish*nprim_max;
        reg_cei[ip] = coeff_exp[ish_ip];
    }
    for (int jp = 0; jp < npj; jp++){
        const int jsh_jp = jp + jsh*nprim_max;
        reg_cej[jp] = coeff_exp[jsh_jp];
    }
    // Cache per-(ip,jp) terms to avoid repeated expensive exp/div computations
    DataType reg_cicj[npi*npj];
    DataType reg_inv_aij[npi*npj];
#pragma unroll
    for (int ip = 0; ip < npi; ip++){
        for (int jp = 0; jp < npj; jp++){
            const DataType ai = reg_cei[ip].e;
            const DataType aj = reg_cej[jp].e;
            const DataType aij = ai + aj;
            const DataType inv_aij = one / aij;
            const DataType aj_aij = aj * inv_aij;
            const DataType theta_ij = ai * aj_aij;
            const DataType Kab = exp(-theta_ij * rr_ij);
            const DataType ci = reg_cei[ip].c;
            const DataType cj = reg_cej[jp].c;
            const DataType cicj = fac_sym * ci * cj * Kab;
            const int idx = ip + jp*npi;
            reg_cicj[idx] = cicj;
            reg_inv_aij[idx] = inv_aij;
        }
    }

    DataType integral_frag[frag_size] = {zero};
    for (int kp = 0; kp < npk; kp++)
    for (int lp = 0; lp < npl; lp++){
        const int ksh_kp = kp + ksh*nprim_max;
        const int lsh_lp = lp + lsh*nprim_max;
        const DataType2 cek = coeff_exp[ksh_kp];
        const DataType2 cel = coeff_exp[lsh_lp];
        const DataType ak = cek.e;
        const DataType al = cel.e;
        const DataType akl = ak + al;
        const DataType inv_akl = one / akl;
        const DataType al_akl = al * inv_akl;
        const DataType theta_kl = ak * al_akl;
        const DataType Kcd = exp(-theta_kl * rr_kl);
        const DataType ck = cek.c;
        const DataType cl = cel.c;
        const DataType ckcl = ck * cl * Kcd;

        for (int ip = 0; ip < npi; ip++)
        for (int jp = 0; jp < npj; jp++){
            const DataType ai = reg_cei[ip].e;
            const DataType aj = reg_cej[jp].e;
            const DataType aij = ai + aj;
            const int idx = ip + jp*npi;
            const DataType inv_aij = reg_inv_aij[idx];
            const DataType aj_aij = aj * inv_aij;
            const DataType cicj = reg_cicj[idx];
            
            const DataType xij = rjri0 * aj_aij + ri.x;
            const DataType yij = rjri1 * aj_aij + ri.y;
            const DataType zij = rjri2 * aj_aij + ri.z;
            const DataType xkl = rlrk0 * al_akl + rk.x;
            const DataType ykl = rlrk1 * al_akl + rk.y;
            const DataType zkl = rlrk2 * al_akl + rk.z;
            const DataType Rpq[3] = {xij-xkl, yij-ykl, zij-zkl};

            const DataType rr = Rpq[0]*Rpq[0] + Rpq[1]*Rpq[1] + Rpq[2]*Rpq[2];
            const DataType inv_aijkl = one / (aij + akl);
            const DataType theta = aij * akl * inv_aijkl;
            
            DataType rjri_x = (ty == 0 ? rjri0 : (ty == 1 ? rjri1 : rjri2)); 
            DataType Rpq_x =  (ty == 0 ? (xij-xkl) : (ty == 1 ? (yij-ykl) : (zij-zkl)));
            DataType rlrk_x = (ty == 0 ? rlrk0 : (ty == 1 ? rlrk1 : rlrk2));

            DataType *rw = shared_memory + tx;
            DataType *g = shared_memory + nroots * 2 * gx_stride + tx; 

            rys_roots(rr, rw, ty, gx_stride, theta, omega);
            
            DataType g0xyz;
            if (ty == 0) g0xyz = ckcl; 
            if (ty == 1) g0xyz = cicj * inv_aij * inv_akl * sqrt(inv_aijkl);
            
            __syncthreads();
            for (int irys = 0; irys < nroots; irys++){
                DataType rt_aa;
                g0xyz = (ty == 2) ? rw[(irys*2+1) * gx_stride] : g0xyz;
                if (ty < 3){
                    const DataType rt = rw[(irys*2)*gx_stride];
                    rt_aa = rt * inv_aijkl;
                }
                __syncthreads();
                if (ty < 3) g[ty*gx_stride] = g0xyz;

                // TRR
                //for i in range(lij):
                //    trr(i+1,0) = c0 * trr(i,0) + i*b10 * trr(i-1,0)
                //for k in range(lkl):
                //    for i in range(lij+1):
                //        trr(i,k+1) = c0p * trr(i,k) + k*b01 * trr(i,k-1) + i*b00 * trr(i-1,k)
                constexpr int lij = li + lj;
                if constexpr (lij > 0) {
                    if (ty < 3){
                        const DataType rt_aij = rt_aa * akl;
                        const DataType b10 = half * inv_aij * (one - rt_aij);

                        const int _ix = ty;
                        DataType *gx = g + _ix * gx_stride;

                        // gx(0,n+1) = c0*gx(0,n) + n*b10*gx(0,n-1)
                        const DataType Rpa = aj_aij * rjri_x;
                        const DataType c0x = Rpa - rt_aij * Rpq_x;
                        DataType s0x, s1x, s2x;
                        s0x = g0xyz;
                        s1x = c0x * s0x;
                        gx[stride_i] = s1x;

                        for (int i = 1; i < lij; ++i) {
                            const DataType i_b10 = i * b10;  // Pre-compute to reduce FLOPs
                            s2x = c0x * s1x + i_b10 * s0x;
                            gx[i*stride_i + stride_i] = s2x;
                            s0x = s1x;
                            s1x = s2x;
                        }
                    }
                }
                                
                constexpr int lkl = lk + ll;
                if constexpr (lkl > 0) {
                    if (ty < 3){
                        const DataType rt_akl = rt_aa * aij;
                        const DataType b00 = half * rt_aa;
                        const DataType b01 = half * inv_akl * (one - rt_akl);

                        const int _ix = ty;
                        DataType *gx = g + _ix * gx_stride;
                        
                        const DataType Rqc = al_akl * rlrk_x;
                        const DataType cpx = Rqc + rt_akl * Rpq_x;
                        
                        //  trr(0,1) = c0p * trr(0,0)
                        DataType s0x, s1x, s2x;
                        s0x = g0xyz;
                        s1x = cpx * s0x;
                        gx[stride_k] = s1x;
                        
                        // trr(0,k+1) = cp * trr(0,k) + k*b01 * trr(0,k-1)
#pragma unroll
                        for (int k = 1; k < lkl; ++k) {
                            const DataType k_b01 = k * b01;  // Pre-compute to reduce FLOPs
                            s2x = cpx*s1x + k_b01*s0x;
                            gx[k*stride_k + stride_k] = s2x;
                            s0x = s1x;
                            s1x = s2x;
                        }
#pragma unroll
                        for (int i = 1; i < lij+1; i++){
                            //for i in range(1, lij+1):
                            //    trr(i,1) = c0p * trr(i,0) + i*b00 * trr(i-1,0)
                            const DataType ib00 = i * b00;
                            const int i_off = i * stride_i;
                            const int i_off_minus = i_off - stride_i;
                            const int i_off_plus_k = i_off + stride_k;
                            s0x = gx[i_off];
                            s1x = cpx * s0x;
                            s1x += ib00 * gx[i_off_minus];
                            gx[i_off_plus_k] = s1x;

                            //for k in range(1, lkl):
                            //    for i in range(lij+1):
                            //        trr(i,k+1) = cp * trr(i,k) + k*b01 * trr(i,k-1) + i*b00 * trr(i-1,k)
                            
                            for (int k = 1; k < lkl; ++k) {
                                const DataType k_b01 = k * b01;  // Pre-compute to reduce FLOPs
                                s2x = cpx*s1x + k_b01*s0x;
                                s2x += ib00 * gx[i_off_minus + k*stride_k];
                                gx[i_off_plus_k + k*stride_k] = s2x;
                                s0x = s1x;
                                s1x = s2x;
                            }
                        }
                    }
                }
                
                const int _ix = ty;
                DataType *gx = g + _ix * gx_stride;
                // hrr
                // g(i,j+1) = rirj * g(i,j) +  g(i+1,j)
                // g(...,k,l+1) = rkrl * g(...,k,l) + g(...,k+1,l)
                if constexpr (lj > 0) {
                    constexpr int stride_j_i = stride_j - stride_i;
                    if (ty < 3){
#pragma unroll
                        for (int kl = 0; kl < lkl+1; kl++){
                            const int kl_off = kl*stride_k;
                            const int ijkl0 = kl_off + lij*stride_i;
                            for (int j = 0; j < lj; ++j) {
                                DataType s0x, s1x;
                                const int jkl_off = kl_off + j*stride_j;
                                int ijkl = ijkl0 + j*stride_j_i;
                                s1x = gx[ijkl];
                                for (ijkl-=stride_i; ijkl >= jkl_off; ijkl-=stride_i) {
                                    s0x = gx[ijkl];
                                    gx[ijkl + stride_j] = s1x - rjri_x * s0x;  // Reordered for consistency
                                    s1x = s0x;
                                }
                            }
                        }
                    }
                }

                if constexpr (ll > 0) {
                    constexpr int li1xlj1 = (li+1)*(lj+1);
                    constexpr int stride_l_k = stride_l - stride_k;
                    if (ty < 3){
#pragma unroll
                        for (int ij = 0; ij < li1xlj1; ij++){
                            const int ij_off = ij*stride_i;
                            const int ijl = lkl*stride_k + ij_off;
                            for (int l = 0; l < ll; ++l) {
                                const int lstride_l = l*stride_l;
                                int ijkl = ijl + l*stride_l_k;
                                DataType s0x, s1x;
                                s1x = gx[ijkl];
                                for (ijkl-=stride_k; ijkl >= lstride_l; ijkl-=stride_k) {
                                    s0x = gx[ijkl];
                                    gx[ijkl + stride_l] = s1x - rlrk_x * s0x;
                                    s1x = s0x;
                                }
                            }
                        }
                    }
                }
                __syncthreads();
                
                if (ty < nt_active) {
                const int idx_off = t_k * fragk * nfl + t_l * fragl;
#pragma unroll
                for (int reg_i = 0; reg_i < fragi; reg_i++){
                    const int i = t_i * fragi + reg_i;
                    for (int reg_j = 0; reg_j < fragj; reg_j++){
                        const int j = t_j * fragj + reg_j;  
                        const uint32_t addr_ij = j_idx[j] + i_idx[i];
                        const uint32_t ij_x    =  addr_ij        & 0x3FF;      // 10 low-order bits
                        const uint32_t ij_y    = (addr_ij >> 10) & 0x3FF;      // next 10 bits
                        const uint32_t ij_z    = (addr_ij >> 20) & 0x3FF;      // next 10 bits

                        int integral_off = reg_i * tstride_i + reg_j * tstride_j;
                        for (int reg_k = 0; reg_k < fragk; reg_k++){
                            for (int reg_l = 0; reg_l < fragl; reg_l++){
                                const int kl = reg_l + reg_k * fragl;
                                const int addrx = ij_x * g_stride + kl_idx[kl];
                                const int addry = ij_y * g_stride + kl_idy[kl];
                                const int addrz = ij_z * g_stride + kl_idz[kl];
                                integral_frag[integral_off + reg_l*tstride_l] += g[addrx] * g[addry] * g[addrz];
                            }
                            integral_off += tstride_k;
                        }
                    }
                }
                }
            }
        }
    }
    
    const int nao = ao_loc[nbas];
    
    const int i0 = ao_loc[ish];
    const int j0 = ao_loc[jsh];
    const int k0 = ao_loc[ksh];
    const int l0 = ao_loc[lsh];

    DataType *smem = shared_memory + tx;
    const bool ty_active = (ty < nt_active);
    for (int i_dm = 0; i_dm < n_dm; ++i_dm) {
        // ijkl, ij -> kl
        constexpr int ntij = nti*ntj;
        DataType vj_lk[fragk*fragl] = {zero};
        if (do_j && ty_active){
            const int dm_offset = (i0+t_i*fragi) + (j0+t_j*fragj)*nao;
            DataType *dm_ptr = dm + dm_offset;
#pragma unroll
            for (int i = 0; i < fragi; i++){
                for (int j = 0; j < fragj; j++){
                    const int offset = i + j*nao;
                    DataType dm_ij = __ldg(dm_ptr + offset);
                    int off = i * tstride_i + j * tstride_j;
                    for (int k = 0; k < fragk; k++){
                        for (int l = 0; l < fragl; l++){
                            vj_lk[l + k*fragl] += integral_frag[off + l*tstride_l] * dm_ij;
                        }
                        off += tstride_k;
                    }
                }
            }
        }
        if constexpr(ntij > 1) __syncthreads();
        if (do_j && ty_active){
            const int t_ij = t_i + nti * t_j;
            const int t_kl = t_k * fragk + t_l * fragl * nfk;
            constexpr int smem_kstride = smem_stride;
            constexpr int smem_lstride = smem_stride * nfk;
            DataType* smem_ptr = smem + (t_ij * nfkl + t_kl) * smem_stride;
            const int vj_offset = (l0+t_l*fragl)*nao + (k0+t_k*fragk);
            double *vj_ptr = vj + vj_offset;
#pragma unroll
            for (int k = 0; k < fragk; k++){
                for (int l = 0; l < fragl; l++){
                    if constexpr(ntij > 1){
                        smem_ptr[k*smem_kstride + l*smem_lstride] = vj_lk[l + k*fragl];
                    } else {
                        const int offset = l*nao + k;
                        atomicAdd(vj_ptr + offset, (double)vj_lk[l + k*fragl]);
                    }
                }
            }
        }
        
        if constexpr(do_j && ntij > 1){
            __syncthreads();
            const int vj_offset = l0*nao + k0;
            double *vj_ptr = vj + vj_offset;
            constexpr int stride = nfkl * smem_stride;
            for (int kl = ty; kl < nfkl; kl += nthreads_per_sq){
                DataType vj_tmp = zero;
                const int off = kl * smem_stride;
                for (int m = 0; m < ntij; m++){
                    vj_tmp += smem[off + m*stride];
                }
                const int l = kl / nfk;
                const int k = kl % nfk;
                const int offset = l*nao + k;
                atomicAdd(vj_ptr + offset, (double)vj_tmp);
            }
        }
        
        // ijkl, kl -> ij
        constexpr int ntkl = ntk*ntl;
        DataType dm_kl_cache[fragk*fragl];
        if (do_j && ty_active){
            const int dm_offset = (l0+t_l*fragl)*nao + (k0+t_k*fragk);
            DataType *dm_ptr = dm + dm_offset;
#pragma unroll
            for (int l = 0; l < fragl; l++){
                for (int k = 0; k < fragk; k++){
                    dm_kl_cache[k + l*fragk] = __ldg(dm_ptr + k);
                }
                dm_ptr += nao;
            }
        }
        if constexpr(ntkl > 1) __syncthreads();
        if (do_j && ty_active){
            const int t_kl = t_k + ntk * t_l;
            const int t_ij = t_i * fragi + t_j * fragj * nfi;
            const int smem_off = (t_ij + t_kl * nfij) * smem_stride;
            const int vj_offset = (j0+t_j*fragj)*nao + (i0+t_i*fragi);
            double *vj_ptr = vj + vj_offset;
#pragma unroll
            for (int i = 0; i < fragi; i++){
            for (int j = 0; j < fragj; j++){
                DataType vj_ji = zero;
                int integral_off = i * tstride_i + j * tstride_j;
                for (int l = 0; l < fragl; l++){
                    for (int k = 0; k < fragk; k++){
                        vj_ji += integral_frag[integral_off + k*tstride_k] * dm_kl_cache[k + l*fragk];
                    }
                    integral_off += tstride_l;
                }
                if constexpr(ntkl > 1){
                    const int ij = i + j * nfi;
                    smem[ij * smem_stride + smem_off] = vj_ji;
                } else {
                    const int offset = j*nao + i;
                    atomicAdd(vj_ptr + offset, (double)vj_ji);
                }
            }}
        }
        
        if constexpr(do_j && ntkl > 1){
            __syncthreads();
            const int vj_offset = j0*nao + i0;
            double *vj_ptr = vj + vj_offset;
            constexpr int stride = nfij * smem_stride;
            for (int ij = ty; ij < nfij; ij += nthreads_per_sq){
                DataType vj_tmp = zero;
                const int off = ij * smem_stride;
                for (int m = 0; m < ntkl; m++){
                    vj_tmp += smem[off + m*stride];
                }
                const int j = ij / nfi;
                const int i = ij % nfi;
                const int offset = j*nao + i;
                atomicAdd(vj_ptr + offset, (double)vj_tmp);
            }
        }

        // ijkl, jl -> ik
        constexpr int ntjl = ntj*ntl;
        DataType dm_jl_cache[fragj*fragl];
        if (do_k && ty_active){
            const int dm_offset = (j0+t_j*fragj)*nao + (l0+t_l*fragl);
            DataType *dm_ptr = dm + dm_offset;
#pragma unroll
            for (int j = 0; j < fragj; j++){
                for (int l = 0; l < fragl; l++){
                    dm_jl_cache[l + j*fragl] = __ldg(dm_ptr + l);
                }
                dm_ptr += nao;
            }
        }
        if constexpr(ntjl > 1) __syncthreads();
        if (do_k && ty_active){
            const int vk_offset = (i0+t_i*fragi)*nao + (k0+t_k*fragk);
            double *vk_ptr = vk + vk_offset;
            const int t_jl = t_j + ntj * t_l;
            const int t_ik = t_i * fragi * nfk + t_k * fragk;
            const int smem_off = (t_jl * nfik + t_ik) * smem_stride;
#pragma unroll
            for (int i = 0; i < fragi; i++){
                for (int k = 0; k < fragk; k++){
                    DataType vk_ik = zero;
                    int integral_off = i * tstride_i + k * tstride_k;
                    for (int j = 0; j < fragj; j++){
                        for (int l = 0; l < fragl; l++){
                            vk_ik += integral_frag[integral_off + l*tstride_l] * dm_jl_cache[l + j*fragl];
                        }
                        integral_off += tstride_j;
                    }
                    if constexpr (ntjl > 1){
                        const int ik = i*nfk + k;
                        smem[ik * smem_stride + smem_off] = vk_ik;
                    } else {
                        const int offset = i*nao + k;
                        atomicAdd(vk_ptr + offset, (double)vk_ik);
                    }
                }
            }
        }
        
        if constexpr(do_k && ntjl > 1){
            constexpr int stride = nfik * smem_stride;
            const int vk_offset = i0*nao + k0;
            double *vk_ptr = vk + vk_offset;
            __syncthreads();
            for (int ik = ty; ik < nfik; ik+=nthreads_per_sq){
                DataType vk_tmp = zero;
                const int off = ik * smem_stride;
                for (int m = 0; m < ntjl; m++){
                    vk_tmp += smem[off + m*stride];
                }
                const int k = ik % nfk;
                const int i = ik / nfk;
                const int offset = i*nao + k;
                atomicAdd(vk_ptr + offset, (double)vk_tmp);
            }
        }

        // ijkl, jk -> il
        constexpr int ntjk = ntj*ntk;
        DataType dm_jk_cache[fragj*fragk];
        if (do_k && ty_active){
            const int dm_offset = (j0+t_j*fragj)*nao + (k0+t_k*fragk);
            DataType *dm_ptr = dm + dm_offset;
#pragma unroll
            for (int j = 0; j < fragj; j++){
                for (int k = 0; k < fragk; k++){
                    dm_jk_cache[k + j*fragk] = __ldg(dm_ptr + k);
                }
                dm_ptr += nao;
            }
        }
        if constexpr(ntjk > 1) __syncthreads();
        if (do_k && ty_active){
            const int t_jk = t_j + ntj * t_k;
            const int t_il = t_i * fragi * nfl + t_l * fragl;
            const int smem_off = (t_jk * nfil + t_il) * smem_stride;
            const int vk_offset = (i0+t_i*fragi)*nao + (l0+t_l*fragl);
            double *vk_ptr = vk + vk_offset;
#pragma unroll
            for (int i = 0; i < fragi; i++){
                for (int l = 0; l < fragl; l++){
                    DataType vk_il = zero;
                    int integral_off = i * tstride_i + l * tstride_l;
                    for (int j = 0; j < fragj; j++){
                        for (int k = 0; k < fragk; k++){
                            vk_il += integral_frag[integral_off + k*tstride_k] * dm_jk_cache[k + j*fragk];
                        }
                        integral_off += tstride_j;
                    }
                    if constexpr (ntjk > 1){
                        const int il = i * nfl + l;
                        smem[il * smem_stride + smem_off] = vk_il;
                    } else {
                        const int offset = i*nao + l;
                        atomicAdd(vk_ptr + offset, (double)vk_il);
                    }
                }
            }
        }

        if constexpr(do_k && ntjk > 1){
            __syncthreads();
            const int vk_offset = i0*nao + l0;
            double *vk_ptr = vk + vk_offset;
            for (int il = ty; il < nfil; il += nthreads_per_sq){
                DataType vk_tmp = zero;
                constexpr int stride = nfil * smem_stride;
                const int off = il * smem_stride;
                for (int m = 0; m < ntjk; m++){
                    vk_tmp += smem[off + m*stride];
                }
                const int l = il % nfl;
                const int i = il / nfl;
                const int offset = i*nao + l;
                atomicAdd(vk_ptr + offset, (double)vk_tmp);
            }
        }

        // ijkl, il -> jk
        constexpr int ntil = nti*ntl;
        DataType dm_il_cache[fragi*fragl];
        if (do_k && ty_active){
            const int dm_offset = (i0+t_i*fragi)*nao + (l0+t_l*fragl);
            DataType *dm_ptr = dm + dm_offset;
            for (int i = 0; i < fragi; i++){
                for (int l = 0; l < fragl; l++){
                    dm_il_cache[l + i*fragl] = __ldg(dm_ptr + l);
                }
                dm_ptr += nao;
            }
        }
        
        if constexpr(ntil > 1) __syncthreads();
        if (do_k && ty_active){
            const int t_il = t_l + ntl * t_i;
            const int t_jk = t_j * fragj * nfk + t_k * fragk;
            const int smem_off = (t_jk + t_il * nfjk) * smem_stride;
            const int vk_offset = (j0+t_j*fragj)*nao + (k0+t_k*fragk);
            double *vk_ptr = vk + vk_offset;
#pragma unroll
            for (int j = 0; j < fragj; j++){
                for (int k = 0; k < fragk; k++){
                    DataType vk_jk = zero;
                    int integral_off = j * tstride_j + k * tstride_k;
                    for (int i = 0; i < fragi; i++){
                        for (int l = 0; l < fragl; l++){
                            vk_jk += integral_frag[integral_off + l*tstride_l] * dm_il_cache[l + i*fragl];
                        }
                        integral_off += tstride_i;
                    }
                    if constexpr(ntil > 1){
                        const int jk = j * nfk + k;
                        smem[jk * smem_stride + smem_off] = vk_jk;
                    } else {
                        const int offset = j*nao + k;
                        atomicAdd(vk_ptr + offset, (double)vk_jk);
                    }
                }
            }
        }
        if constexpr(do_k && ntil > 1){
            __syncthreads();
            const int vk_offset = j0*nao + k0;
            double *vk_ptr = vk + vk_offset;
            constexpr int stride = nfjk * smem_stride;
            for (int jk = ty; jk < nfjk; jk += nthreads_per_sq){
                DataType vk_tmp = zero;
                const int off = jk * smem_stride;
                for (int m = 0; m < ntil; m++){
                    vk_tmp += smem[off + m*stride];
                }
                const int k = jk % nfk;
                const int j = jk / nfk;
                const int offset = j*nao + k;
                atomicAdd(vk_ptr + offset, (double)vk_tmp);
            }
        }

        // ijkl, ik -> jl
        constexpr int ntik = nti*ntk;
        DataType vk_jl[fragj*fragl] = {zero};
        if (do_k && ty_active){
            const int dm_offset = (i0+t_i*fragi)*nao + (k0+t_k*fragk);
            DataType *dm_ptr = dm + dm_offset;
#pragma unroll
            for (int i = 0; i < fragi; i++){
                for (int k = 0; k < fragk; k++){
                    const int offset = i*nao + k;
                    DataType dm_ik = __ldg(dm_ptr + offset);
                    int integral_off = i * tstride_i + k * tstride_k;
                    for (int j = 0; j < fragj; j++){
                        for (int l = 0; l < fragl; l++){
                            vk_jl[l + j*fragl] += integral_frag[integral_off + l*tstride_l] * dm_ik;
                        }
                        integral_off += tstride_j;
                    }
                }
            }
        }

        if constexpr(ntik > 1) __syncthreads();
        if (do_k && ty_active){
            const int t_ik = t_i + nti * t_k;
            const int t_jl = t_j * fragj * nfl + t_l * fragl;
            const int smem_off = (t_jl + t_ik * nfjl) * smem_stride;
            const int vk_offset = (j0+t_j*fragj)*nao + (l0+t_l*fragl);
            double *vk_ptr = vk + vk_offset;
            for (int j = 0; j < fragj; j++){
                for (int l = 0; l < fragl; l++){
                    if constexpr(ntik > 1){
                        const int jl = j * nfl + l;
                        smem[jl * smem_stride + smem_off] = vk_jl[l + j*fragl];
                    } else {
                        const int offset = j*nao + l;
                        atomicAdd(vk_ptr + offset, (double)vk_jl[l + j*fragl]);
                    }
                }
            }
        }

        if constexpr(do_k && ntik > 1){
            __syncthreads();
            const int vk_offset = j0*nao + l0;
            double *vk_ptr = vk + vk_offset;
            constexpr int stride = nfjl * smem_stride;
            for (int jl = ty; jl < nfjl; jl+=nthreads_per_sq){
                DataType vk_tmp = zero;
                const int off = jl * smem_stride;
                for (int m = 0; m < ntik; m++){
                    vk_tmp += smem[off + m*stride];
                }
                const int l = jl % nfl;
                const int j = jl / nfl;
                const int offset = j*nao + l;
                atomicAdd(vk_ptr + offset, (double)vk_tmp);
            }
        }
        
        const int nao2 = nao * nao;
        dm += nao2;
        if constexpr(do_j) vj += nao2;
        if constexpr(do_k) vk += nao2;
    }
}
