#include "hip/hip_runtime.h"
/*
# Copyright 2025 ByteDance Inc.
#
# Licensed under the Apache License, Version 2.0 (the "License");
# you may not use this file except in compliance with the License.
# You may obtain a copy of the License at
#
#     http://www.apache.org/licenses/LICENSE-2.0
#
# Unless required by applicable law or agreed to in writing, software
# distributed under the License is distributed on an "AS IS" BASIS,
# WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
# See the License for the specific language governing permissions and
# limitations under the License.
#
*/

// Portions of this file adapted from GPU4PySCF v1.4 (https://github.com/pyscf/gpu4pyscf)
// Copyright 2025 PySCF developer.
// Licensed under the Apache License, Version 2.0.

// 2*pi**2.5
constexpr DataType PI_FAC = 34.98683665524972497;
constexpr DataType half = .5;
constexpr DataType one = 1.0;
constexpr DataType zero = 0.0;
constexpr int nprim_max = 16;

extern "C" __global__
void rys_jk(const int nbas,  
        const int* __restrict__ ao_loc, 
        const DataType* __restrict__ coords,
        const DataType* __restrict__ exponents, 
        const DataType* __restrict__ coeffs,
        DataType* dm, 
        double* vj, 
        double* vk,
        const DataType omega,
        const int4* __restrict__ shl_quartet_idx, 
        const int ntasks)
{
    if (ntasks == 0) return;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int task_id = blockIdx.x * blockDim.x + threadIdx.x;

    constexpr int nfi = (li+1)*(li+2)/2;
    constexpr int nfj = (lj+1)*(lj+2)/2;
    constexpr int nfk = (lk+1)*(lk+2)/2;
    constexpr int nfl = (ll+1)*(ll+2)/2;

    constexpr int nfij = nfi*nfj;
    constexpr int nfkl = nfk*nfl;
    constexpr int nfik = nfi*nfk;
    constexpr int nfil = nfi*nfl;
    constexpr int nfjk = nfj*nfk;
    constexpr int nfjl = nfj*nfl;

    constexpr int nti = (nfi + fragi - 1) / fragi;
    constexpr int ntj = (nfj + fragj - 1) / fragj;
    constexpr int ntk = (nfk + fragk - 1) / fragk;
    constexpr int ntl = (nfl + fragl - 1) / fragl;
    constexpr int nt_active = nti * ntj * ntk * ntl;

    constexpr int tstride_l = 1;
    constexpr int tstride_k = fragl;
    constexpr int tstride_j = fragl * fragk;
    constexpr int tstride_i = fragl * fragk * fragj;
    constexpr int frag_size = fragi*fragj*fragk*fragl;

    const int t_i = (ty % nti);
    const int t_j = (ty / nti % ntj);
    const int t_k = (ty / (nti*ntj) % ntk);
    const int t_l = (ty / (nti*ntj*ntk));

    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    constexpr int gx_stride = nsq_per_block | 1; // reduce bank conflict
    constexpr int g_stride = 3 * gx_stride;

    // shape of g, (gsize, 3, nsq_per_block)
    constexpr int stride_i = g_stride;
    constexpr int stride_j = stride_i * (li+1);
    constexpr int stride_k = stride_j * (lj+1);
    constexpr int stride_l = stride_k * (lk+1);

    // shared memory buffer will be fragmented into three parts
    extern __shared__ DataType shared_memory[];

    __shared__ int kl_idx[nfkl], kl_idy[nfkl], kl_idz[nfkl];
#pragma unroll
    for (int kl = tid; kl < nfkl; kl += threads){
        const int l = kl % nfl;
        const int k = kl / nfl;
        const uint32_t addr = k_idx[k] + l_idx[l];
        const uint32_t kl_x =  addr        & 0x3FF;      // 10 low-order bits
        const uint32_t kl_y = (addr >> 10) & 0x3FF;      // next 10 bits
        const uint32_t kl_z = (addr >> 20) & 0x3FF;      // next 10 bits
        kl_idx[kl] = kl_x * g_stride;
        kl_idy[kl] = kl_y * g_stride + gx_stride;
        kl_idz[kl] = kl_z * g_stride + 2*gx_stride;
    }

    const int4 sq = shl_quartet_idx[task_id];
    const bool active = (task_id < ntasks);
    const int ish = active ? sq.x : 0;
    const int jsh = active ? sq.y : 0;
    const int ksh = active ? sq.z : 0;
    const int lsh = active ? sq.w : 0;

    DataType fac_sym = active ? PI_FAC : zero;
    if (ish == jsh) fac_sym *= half;
    if (ksh == lsh) fac_sym *= half;
    if (ish*nbas+jsh == ksh*nbas+lsh) fac_sym *= half;

    const DataType rij0 = coords[3*jsh] - coords[3*ish];
    const DataType rij1 = coords[3*jsh+1] - coords[3*ish+1];
    const DataType rij2 = coords[3*jsh+2] - coords[3*ish+2];
    const DataType rjri[3] = {rij0, rij1, rij2};
    const DataType rr_ij = rjri[0]*rjri[0] + rjri[1]*rjri[1] + rjri[2]*rjri[2];
    const DataType rkl0 = coords[3*lsh] - coords[3*ksh];
    const DataType rkl1 = coords[3*lsh+1] - coords[3*ksh+1];
    const DataType rkl2 = coords[3*lsh+2] - coords[3*ksh+2];
    const DataType rlrk[3] = {rkl0, rkl1, rkl2};
    const DataType rr_kl = rlrk[0]*rlrk[0] + rlrk[1]*rlrk[1] + rlrk[2]*rlrk[2];

    DataType integral_frag[frag_size] = {0.0};
    for (int kp = 0; kp < npk; kp++)
    for (int lp = 0; lp < npl; lp++){
        const int ksh_kp = kp + ksh*nprim_max;
        const int lsh_lp = lp + lsh*nprim_max;
        const DataType ak = __ldg(exponents + ksh_kp);
        const DataType al = __ldg(exponents + lsh_lp);
        const DataType akl = ak + al;
        const DataType al_akl = al / akl;
        const DataType theta_kl = ak * al_akl;
        const DataType Kcd = exp(-theta_kl * rr_kl);
        const DataType ck = __ldg(coeffs + ksh_kp);
        const DataType cl = __ldg(coeffs + lsh_lp);
        const DataType ckcl = ck * cl * Kcd;
        for (int ip = 0; ip < npi; ip++)
        for (int jp = 0; jp < npj; jp++){
            const int ish_ip = ip + ish*nprim_max;
            const int jsh_jp = jp + jsh*nprim_max;
            const DataType ai = __ldg(exponents + ish_ip);
            const DataType aj = __ldg(exponents + jsh_jp);
            const DataType aij = ai + aj;
            const DataType aj_aij = aj / aij;
            
            const DataType theta_ij = ai * aj_aij;
            const DataType Kab = exp(-theta_ij * rr_ij);
            const DataType ci = __ldg(coeffs + ish_ip);
            const DataType cj = __ldg(coeffs + jsh_jp);
            const DataType cicj = fac_sym * ci * cj * Kab;
            
            const DataType xij = rjri[0] * aj_aij + __ldg(coords + 3*ish);
            const DataType yij = rjri[1] * aj_aij + __ldg(coords + 3*ish+1);
            const DataType zij = rjri[2] * aj_aij + __ldg(coords + 3*ish+2);
            const DataType xkl = rlrk[0] * al_akl + __ldg(coords + 3*ksh);
            const DataType ykl = rlrk[1] * al_akl + __ldg(coords + 3*ksh+1);
            const DataType zkl = rlrk[2] * al_akl + __ldg(coords + 3*ksh+2);
            const DataType Rpq[3] = {xij-xkl, yij-ykl, zij-zkl};

            const DataType rr = Rpq[0]*Rpq[0] + Rpq[1]*Rpq[1] + Rpq[2]*Rpq[2];
            const DataType theta = aij * akl / (aij + akl);
            
            DataType rjri_x = (ty == 0 ? rjri[0] : (ty == 1 ? rjri[1] : rjri[2])); 
            DataType Rpq_x =  (ty == 0 ? Rpq[0] : (ty == 1 ? Rpq[1] : Rpq[2]));
            DataType rlrk_x = (ty == 0 ? rlrk[0] : (ty == 1 ? rlrk[1] : rlrk[2]));

            DataType *rw = shared_memory + tx;
            DataType *g = shared_memory +  nroots * 2 * gx_stride + tx; 

            rys_roots(rr, rw, ty, gx_stride, theta, omega);
            
            DataType g0xyz;
            if (ty == 0) g0xyz = ckcl; 
            if (ty == 1) g0xyz = cicj / (aij*akl*sqrt(aij+akl));
            
            __syncthreads();
            for (int irys = 0; irys < nroots; irys++){
                DataType rt_aa;
                if (ty == 2) g0xyz = rw[(irys*2+1) * gx_stride];
                if (ty < 3){
                    const DataType rt = rw[(irys*2)*gx_stride];
                    rt_aa = rt / (aij + akl);
                }
                __syncthreads();
                if (ty < 3) g[ty*gx_stride] = g0xyz;

                // TRR
                //for i in range(lij):
                //    trr(i+1,0) = c0 * trr(i,0) + i*b10 * trr(i-1,0)
                //for k in range(lkl):
                //    for i in range(lij+1):
                //        trr(i,k+1) = c0p * trr(i,k) + k*b01 * trr(i,k-1) + i*b00 * trr(i-1,k)
                constexpr int lij = li + lj;
                if constexpr (lij > 0) {
                    if (ty < 3){
                        const DataType rt_aij = rt_aa * akl;
                        const DataType b10 = half/aij * (one - rt_aij);

                        const int _ix = ty;
                        DataType *gx = g + _ix * gx_stride;

                        // gx(0,n+1) = c0*gx(0,n) + n*b10*gx(0,n-1)
                        const DataType Rpa = aj_aij * rjri_x;
                        const DataType c0x = Rpa - rt_aij * Rpq_x;
                        DataType s0x, s1x, s2x;
                        s0x = g0xyz;
                        s1x = c0x * s0x;
                        gx[stride_i] = s1x;

                        for (int i = 1; i < lij; ++i) {
                            s2x = c0x * s1x + i * b10 * s0x;
                            gx[i*stride_i + stride_i] = s2x;
                            s0x = s1x;
                            s1x = s2x;
                        }
                    }
                }

                constexpr int lkl = lk + ll;
                if constexpr (lkl > 0) {
                    if (ty < 3){
                        const DataType rt_akl = rt_aa * aij;
                        const DataType b00 = half * rt_aa;
                        const DataType b01 = half/akl * (one - rt_akl);

                        const int _ix = ty;
                        DataType *gx = g + _ix * gx_stride;

                        const DataType Rqc = al_akl * rlrk_x; 
                        const DataType cpx = Rqc + rt_akl * Rpq_x;
                        
                        //  trr(0,1) = c0p * trr(0,0)
                        DataType s0x, s1x, s2x;
                        s0x = g0xyz;
                        s1x = cpx * s0x;
                        gx[stride_k] = s1x;
                        
                        // trr(0,k+1) = cp * trr(0,k) + k*b01 * trr(0,k-1)
#pragma unroll
                        for (int k = 1; k < lkl; ++k) {
                            s2x = cpx*s1x + k*b01*s0x;
                            gx[k*stride_k + stride_k] = s2x;
                            s0x = s1x;
                            s1x = s2x;
                        }
#pragma unroll
                        for (int i = 1; i < lij+1; i++){
                            //for i in range(1, lij+1):
                            //    trr(i,1) = c0p * trr(i,0) + i*b00 * trr(i-1,0)
                            const DataType ib00 = i * b00;
                            const int i_off = i * stride_i;
                            const int i_off_minus = i_off - stride_i;
                            const int i_off_plus_k = i_off + stride_k;
                            s0x = gx[i_off];
                            s1x = cpx * s0x;
                            s1x += ib00 * gx[i_off_minus];
                            gx[i_off_plus_k] = s1x;

                            DataType kb01 = zero;
                            //for k in range(1, lkl):
                            //    for i in range(lij+1):
                            //        trr(i,k+1) = cp * trr(i,k) + k*b01 * trr(i,k-1) + i*b00 * trr(i-1,k)
                            for (int k = 1; k < lkl; ++k) {
                                kb01 += b01;
                                s2x = cpx*s1x + kb01*s0x;
                                s2x += ib00 * gx[i_off_minus + k*stride_k];
                                gx[i_off_plus_k + k*stride_k] = s2x;
                                s0x = s1x;
                                s1x = s2x;
                            }
                        }
                    }
                }
                
                const int _ix = ty;
                DataType *gx = g + _ix * gx_stride;
                // hrr
                // g(i,j+1) = rirj * g(i,j) +  g(i+1,j)
                // g(...,k,l+1) = rkrl * g(...,k,l) + g(...,k+1,l)
                if constexpr (lj > 0) {
                    constexpr int stride_j_i = stride_j - stride_i;
                    if (ty < 3){
#pragma unroll
                        for (int kl = 0; kl < lkl+1; kl++){
                            const int kl_off = kl*stride_k;
                            const int ijkl0 = kl_off + lij*stride_i;
                            for (int j = 0; j < lj; ++j) {
                                DataType s0x, s1x;
                                const int jkl_off = kl_off + j*stride_j;
                                int ijkl = ijkl0 + j*stride_j_i;
                                s1x = gx[ijkl];
                                for (ijkl-=stride_i; ijkl >= jkl_off; ijkl-=stride_i) {
                                    s0x = gx[ijkl];
                                    gx[ijkl + stride_j] = s1x - s0x * rjri_x;
                                    s1x = s0x;
                                }
                            }
                        }
                    }
                }

                if constexpr (ll > 0) {
                    constexpr int li1xlj1 = (li+1)*(lj+1);
                    constexpr int stride_l_k = stride_l - stride_k;
                    if (ty < 3){
#pragma unroll
                        for (int ij = 0; ij < li1xlj1; ij++){
                            const int ij_off = ij*stride_i;
                            const int ijl = lkl*stride_k + ij_off;
                            for (int l = 0; l < ll; ++l) {
                                const int lstride_l = l*stride_l;
                                int ijkl = ijl + l*stride_l_k;
                                DataType s0x, s1x;
                                s1x = gx[ijkl];
                                for (ijkl-=stride_k; ijkl >= lstride_l; ijkl-=stride_k) {
                                    s0x = gx[ijkl];
                                    gx[ijkl + stride_l] = s1x - rlrk_x * s0x;
                                    s1x = s0x;
                                }
                            }
                        }
                    }
                }
                __syncthreads();
                if (ty >= nt_active) continue;
                
                const int idx_off = t_k * fragk * nfl + t_l * fragl;
#pragma unroll
                for (int reg_i = 0; reg_i < fragi; reg_i++){
                    const int i = t_i * fragi + reg_i;
                    for (int reg_j = 0; reg_j < fragj; reg_j++){
                        const int j = t_j * fragj + reg_j;  
                        const uint32_t addr_ij = j_idx[j] + i_idx[i];
                        const uint32_t ij_x    =  addr_ij        & 0x3FF;      // 10 low-order bits
                        const uint32_t ij_y    = (addr_ij >> 10) & 0x3FF;      // next 10 bits
                        const uint32_t ij_z    = (addr_ij >> 20) & 0x3FF;      // next 10 bits
                        
                        int integral_off = reg_i * tstride_i + reg_j * tstride_j;
                        for (int reg_k = 0; reg_k < fragk; reg_k++){
                            const int kl_off = reg_k * nfl + idx_off;
                            for (int reg_l = 0; reg_l < fragl; reg_l++){
                                const int kl = kl_off + reg_l;
                                const int addrx = ij_x * g_stride + kl_idx[kl];
                                const int addry = ij_y * g_stride + kl_idy[kl];
                                const int addrz = ij_z * g_stride + kl_idz[kl];
                                integral_frag[integral_off + reg_l*tstride_l] += g[addrx] * g[addry] * g[addrz];
                            }
                            integral_off += tstride_k;
                        }
                    }
                }
            }
        }
    }
    
    const int nao = ao_loc[nbas];
    
    const int i0 = ao_loc[ish];
    const int j0 = ao_loc[jsh];
    const int k0 = ao_loc[ksh];
    const int l0 = ao_loc[lsh];

    DataType *smem = shared_memory + tx;
    constexpr int smem_stride = nsq_per_block | 1;
    for (int i_dm = 0; i_dm < n_dm; ++i_dm) {
        // ijkl, ij -> kl
        constexpr int ntij = nti*ntj;
        if constexpr(ntij > 1) __syncthreads();
        if (do_j && ty < nt_active){
            const int dm_offset = (i0+t_i*fragi) + (j0+t_j*fragj)*nao;
            DataType *dm_ptr = dm + dm_offset;
            DataType vj_lk[fragk*fragl] = {0.0};
#pragma unroll
            for (int i = 0; i < fragi; i++){
                for (int j = 0; j < fragj; j++){
                    const int offset = i + j*nao;
                    DataType dm_ij = __ldg(dm_ptr + offset);
                    int off = i * tstride_i + j * tstride_j;
                    for (int k = 0; k < fragk; k++){
                        for (int l = 0; l < fragl; l++){
                            vj_lk[l + k*fragl] += integral_frag[off + l*tstride_l] * dm_ij;
                        }
                        off += tstride_k;
                    }
                }
            }
            const int t_ij = t_i + nti * t_j;
            const int t_kl = t_k * fragk + t_l * fragl * nfk;
            constexpr int smem_kstride = smem_stride;
            constexpr int smem_lstride = smem_stride * nfk;
            DataType* smem_ptr = smem + (t_ij * nfkl + t_kl) * smem_stride;
            const int vj_offset = (l0+t_l*fragl)*nao + (k0+t_k*fragk);
            double *vj_ptr = vj + vj_offset;
            for (int k = 0; k < fragk; k++){
                for (int l = 0; l < fragl; l++){
                    if constexpr(ntij > 1){
                        smem_ptr[k*smem_kstride + l*smem_lstride] = vj_lk[l + k*fragl];
                    } else {
                        const int offset = l*nao + k;
                        atomicAdd(vj_ptr + offset, (double)vj_lk[l + k*fragl]);
                    }
                }
            }
        }
        
        if constexpr(do_j && ntij > 1){
            __syncthreads();
            const int vj_offset = l0*nao + k0;
            double *vj_ptr = vj + vj_offset;
            constexpr int stride = nfkl * smem_stride;
            for (int kl = ty; kl < nfkl; kl += nthreads_per_sq){
                DataType vj_tmp = 0.0;
                const int off = kl * smem_stride;
                for (int m = 0; m < ntij; m++){
                    vj_tmp += smem[off + m*stride];
                }
                const int l = kl / nfk;
                const int k = kl % nfk;
                const int offset = l*nao + k;
                atomicAdd(vj_ptr + offset, (double)vj_tmp);
            }
        }

        // ijkl, kl -> ij
        constexpr int ntkl = ntk*ntl;
        if constexpr(ntkl > 1) __syncthreads();
        if (do_j && ty < nt_active){
            DataType dm_kl_cache[fragk*fragl];
            const int dm_offset = (l0+t_l*fragl)*nao + (k0+t_k*fragk);
            DataType *dm_ptr = dm + dm_offset;
            for (int l = 0; l < fragl; l++){
                for (int k = 0; k < fragk; k++){
                    dm_kl_cache[k + l*fragk] = __ldg(dm_ptr + k);
                }
                dm_ptr += nao;
            }
            const int t_kl = t_k + ntk * t_l;
            const int t_ij = t_i * fragi + t_j * fragj * nfi;
            const int smem_off = (t_ij + t_kl * nfij) * smem_stride;
            const int vj_offset = (j0+t_j*fragj)*nao + (i0+t_i*fragi);
            double *vj_ptr = vj + vj_offset;
#pragma unroll
            for (int i = 0; i < fragi; i++){
            for (int j = 0; j < fragj; j++){
                DataType vj_ji = zero;
                int integral_off = i * tstride_i + j * tstride_j;
                for (int l = 0; l < fragl; l++){
                    for (int k = 0; k < fragk; k++){
                        vj_ji += integral_frag[integral_off + k*tstride_k] * dm_kl_cache[k + l*fragk];
                    }
                    integral_off += tstride_l;
                }

                if constexpr(ntkl > 1){
                    const int ij = i + j * nfi;
                    smem[ij * smem_stride + smem_off] = vj_ji;
                } else {
                    const int offset = j*nao + i;
                    atomicAdd(vj_ptr + offset, (double)vj_ji);
                }
            }}
        }
        
        if constexpr(do_j && ntkl > 1){
            __syncthreads();
            const int vj_offset = j0*nao + i0;
            double *vj_ptr = vj + vj_offset;
            constexpr int stride = nfij * smem_stride;
            for (int ij = ty; ij < nfij; ij += nthreads_per_sq){
                DataType vj_tmp = 0.0;
                const int off = ij * smem_stride;
                for (int m = 0; m < ntkl; m++){
                    vj_tmp += smem[off + m*stride];
                }
                const int j = ij / nfi;
                const int i = ij % nfi;
                const int offset = j*nao + i;
                atomicAdd(vj_ptr + offset, (double)vj_tmp);
            }
        }

        // ijkl, jl -> ik
        constexpr int ntjl = ntj*ntl;
        if constexpr(ntjl > 1) __syncthreads();
        if (do_k && ty < nt_active){
            const int t_jl = t_j + ntj * t_l;
            const int t_ik = t_i * fragi * nfk + t_k * fragk;
            const int smem_off = (t_jl * nfik + t_ik) * smem_stride;
            const int vk_offset = (i0+t_i*fragi)*nao + (k0+t_k*fragk);
            double *vk_ptr = vk + vk_offset;
            DataType dm_jl_cache[fragj*fragl];
            const int dm_offset = (j0+t_j*fragj)*nao + (l0+t_l*fragl);
            DataType *dm_ptr = dm + dm_offset;
#pragma unroll
            for (int j = 0; j < fragj; j++){
                for (int l = 0; l < fragl; l++){
                    dm_jl_cache[l + j*fragl] = __ldg(dm_ptr + l);
                }
                dm_ptr += nao;
            }
#pragma unroll
            for (int i = 0; i < fragi; i++){
                for (int k = 0; k < fragk; k++){
                    DataType vk_ik = zero;
                    int integral_off = i * tstride_i + k * tstride_k;
                    for (int j = 0; j < fragj; j++){
                        for (int l = 0; l < fragl; l++){
                            vk_ik += integral_frag[integral_off + l*tstride_l] * dm_jl_cache[l + j*fragl];
                        }
                        integral_off += tstride_j;
                    }

                    if constexpr (ntjl > 1){
                        const int ik = i*nfk + k;
                        smem[ik * smem_stride + smem_off] = vk_ik;
                    } else {
                        const int offset = i*nao + k;
                        atomicAdd(vk_ptr + offset, (double)vk_ik);
                    }
                }
            }
        }
        
        if constexpr(do_k && ntjl > 1){
            constexpr int stride = nfik * smem_stride;
            const int vk_offset = i0*nao + k0;
            double *vk_ptr = vk + vk_offset;
            __syncthreads();
            for (int ik = ty; ik < nfik; ik+=nthreads_per_sq){
                DataType vk_tmp = 0.0;
                const int off = ik * smem_stride;
                for (int m = 0; m < ntjl; m++){
                    vk_tmp += smem[off + m*stride];
                }
                const int k = ik % nfk;
                const int i = ik / nfk;
                const int offset = i*nao + k;
                atomicAdd(vk_ptr + offset, (double)vk_tmp);
            }
        }

        // ijkl, jk -> il
        constexpr int ntjk = ntj*ntk;
        if constexpr(ntjk > 1) __syncthreads();
        if (do_k && ty < nt_active){
            DataType dm_jk_cache[fragj*fragk];
            const int dm_offset = (j0+t_j*fragj)*nao + (k0+t_k*fragk);
            DataType *dm_ptr = dm + dm_offset;
#pragma unroll
            for (int j = 0; j < fragj; j++){
                for (int k = 0; k < fragk; k++){
                    dm_jk_cache[k + j*fragk] = __ldg(dm_ptr + k);
                }
                dm_ptr += nao;
            }
            const int t_jk = t_j + ntj * t_k;
            const int t_il = t_i * fragi * nfl + t_l * fragl;
            const int smem_off = (t_jk * nfil + t_il) * smem_stride;
            const int vk_offset = (i0+t_i*fragi)*nao + (l0+t_l*fragl);
            double *vk_ptr = vk + vk_offset;
#pragma unroll
            for (int i = 0; i < fragi; i++){
                for (int l = 0; l < fragl; l++){
                    DataType vk_il = 0.0;
                    int integral_off = i * tstride_i + l * tstride_l;
                    for (int j = 0; j < fragj; j++){
                        for (int k = 0; k < fragk; k++){
                            vk_il += integral_frag[integral_off + k*tstride_k] * dm_jk_cache[k + j*fragk];
                        }
                        integral_off += tstride_j;
                    }

                    if constexpr (ntjk > 1){
                        const int il = i * nfl + l;
                        smem[il * smem_stride + smem_off] = vk_il;
                    } else {
                        const int offset = i*nao + l;
                        atomicAdd(vk_ptr + offset, (double)vk_il);
                    }
                }
            }
        }

        if constexpr(do_k && ntjk > 1){
            __syncthreads();
            const int vk_offset = i0*nao + l0;
            double *vk_ptr = vk + vk_offset;
            for (int il = ty; il < nfil; il += nthreads_per_sq){
                DataType vk_tmp = 0.0;
                constexpr int stride = nfil * smem_stride;
                const int off = il * smem_stride;
                for (int m = 0; m < ntjk; m++){
                    vk_tmp += smem[off + m*stride];
                }
                const int l = il % nfl;
                const int i = il / nfl;
                const int offset = i*nao + l;
                atomicAdd(vk_ptr + offset, (double)vk_tmp);
            }
        }

        // ijkl, il -> jk
        constexpr int ntil = nti*ntl;
        if constexpr(ntil > 1) __syncthreads();
        if (do_k && ty < nt_active){
            DataType dm_il_cache[fragi*fragl];
            const int dm_offset = (i0+t_i*fragi)*nao + (l0+t_l*fragl);
            DataType *dm_ptr = dm + dm_offset;
            for (int i = 0; i < fragi; i++){
                for (int l = 0; l < fragl; l++){
                    dm_il_cache[l + i*fragl] = __ldg(dm_ptr + l);
                }
                dm_ptr += nao;
            }
            const int t_il = t_l + ntl * t_i;
            const int t_jk = t_j * fragj * nfk + t_k * fragk;
            const int smem_off = (t_jk + t_il * nfjk) * smem_stride;
            const int vk_offset = (j0+t_j*fragj)*nao + (k0+t_k*fragk);
            double *vk_ptr = vk + vk_offset;
#pragma unroll
            for (int j = 0; j < fragj; j++){
                for (int k = 0; k < fragk; k++){
                    DataType vk_jk = zero;
                    int integral_off = j * tstride_j + k * tstride_k;
                    for (int i = 0; i < fragi; i++){
                        for (int l = 0; l < fragl; l++){
                            vk_jk += integral_frag[integral_off + l*tstride_l] * dm_il_cache[l + i*fragl];
                        }
                        integral_off += tstride_i;
                    }
                    if constexpr(ntil > 1){
                        const int jk = j * nfk + k;
                        smem[jk * smem_stride + smem_off] = vk_jk;
                    } else {
                        const int offset = j*nao + k;
                        atomicAdd(vk_ptr + offset, (double)vk_jk);
                    }
                }
            }
        }
        if constexpr(do_k && ntil > 1){
            __syncthreads();
            const int vk_offset = j0*nao + k0;
            double *vk_ptr = vk + vk_offset;
            constexpr int stride = nfjk * smem_stride;
            for (int jk = ty; jk < nfjk; jk += nthreads_per_sq){
                DataType vk_tmp = 0.0;
                const int off = jk * smem_stride;
                for (int m = 0; m < ntil; m++){
                    vk_tmp += smem[off + m*stride];
                }
                const int k = jk % nfk;
                const int j = jk / nfk;
                const int offset = j*nao + k;
                atomicAdd(vk_ptr + offset, (double)vk_tmp);
            }
        }

        // ijkl, ik -> jl
        constexpr int ntik = nti*ntk;
        if constexpr(ntik > 1) __syncthreads();
        if (do_k && ty < nt_active){
            const int t_ik = t_i + nti * t_k;
            const int t_jl = t_j * fragj * nfl + t_l * fragl;
            const int dm_offset = (i0+t_i*fragi)*nao + (k0+t_k*fragk);
            DataType *dm_ptr = dm + dm_offset;
            DataType vk_jl[fragj*fragl] = {0.0};
#pragma unroll
            for (int i = 0; i < fragi; i++){
                for (int k = 0; k < fragk; k++){
                    const int offset = i*nao + k;
                    DataType dm_ik = __ldg(dm_ptr + offset);
                    int integral_off = i * tstride_i + k * tstride_k;
                    for (int j = 0; j < fragj; j++){
                        for (int l = 0; l < fragl; l++){
                            vk_jl[l + j*fragl] += integral_frag[integral_off + l*tstride_l] * dm_ik;
                        }
                        integral_off += tstride_j;
                    }
                }
            }
            
            const int smem_off = (t_jl + t_ik * nfjl) * smem_stride;
            const int vk_offset = (j0+t_j*fragj)*nao + (l0+t_l*fragl);
            double *vk_ptr = vk + vk_offset;
            for (int j = 0; j < fragj; j++){
                for (int l = 0; l < fragl; l++){
                    if constexpr(ntik > 1){
                        const int jl = j * nfl + l;
                        smem[jl * smem_stride + smem_off] = vk_jl[l + j*fragl];
                    } else {
                        const int offset = j*nao + l;
                        atomicAdd(vk_ptr + offset, (double)vk_jl[l + j*fragl]);
                    }
                }
            }
        }

        if constexpr(do_k && ntik > 1){
            __syncthreads();
            const int vk_offset = j0*nao + l0;
            double *vk_ptr = vk + vk_offset;
            constexpr int stride = nfjl * smem_stride;
            for (int jl = ty; jl < nfjl; jl+=nthreads_per_sq){
                DataType vk_tmp = 0.0;
                const int off = jl * smem_stride;
                for (int m = 0; m < ntik; m++){
                    vk_tmp += smem[off + m*stride];
                }
                const int l = jl % nfl;
                const int j = jl / nfl;
                const int offset = j*nao + l;
                atomicAdd(vk_ptr + offset, (double)vk_tmp);
            }
        }
        const int nao2 = nao * nao;
        dm += nao2;
        if constexpr(do_j) vj += nao2;
        if constexpr(do_k) vk += nao2;
    }
}
