#include "hip/hip_runtime.h"
/*
# Copyright 2025 ByteDance Inc.
#
# Licensed under the Apache License, Version 2.0 (the "License");
# you may not use this file except in compliance with the License.
# You may obtain a copy of the License at
#
#     http://www.apache.org/licenses/LICENSE-2.0
#
# Unless required by applicable law or agreed to in writing, software
# distributed under the License is distributed on an "AS IS" BASIS,
# WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
# See the License for the specific language governing permissions and
# limitations under the License.
#
*/

// Portions of this file adapted from GPU4PySCF v1.4 (https://github.com/pyscf/gpu4pyscf)
// Copyright 2025 PySCF developer.
// Licensed under the Apache License, Version 2.0.

typedef unsigned long uint64_t;

__forceinline__ __device__
int global_offset(int* batch_head, int val){
    // Calculate the cumulative sum of the count array
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;
    const int lane  = tid & 31;    
    const int warp  = tid >> 5;
    for (int ofs = 1; ofs < 32; ofs <<= 1) {
        int n = __shfl_up_sync(0xffffffff, val, ofs);
        if (lane >= ofs) val += n;                       
    }
    __syncthreads();
    __shared__ int cum_count[threads];
    
    cum_count[tid] = val;

    __shared__ int warp_tot[threads / 32];  
    if (lane == 31) warp_tot[warp] = val;  
    __syncthreads(); 

    if (warp == 0) {
        int warp_val = warp_tot[lane];
#pragma unroll
        for (int ofs = 1; ofs < 32; ofs <<= 1) {
            int n = __shfl_up_sync(0xffffffff, warp_val, ofs);
            if (lane >= ofs) warp_val += n;
        }
        warp_tot[lane] = warp_val;
    }
    __syncthreads();  
    int warp_offset = (warp == 0) ? 0 : warp_tot[warp-1];
    cum_count[tid] = val + warp_offset;
    __syncthreads();

    const int ntasks = cum_count[threads-1];
    if (ntasks == 0) return; 
    
    // Calculate the global offset
    int offset = 0;
    if (tid == 0){
        offset = atomicAdd(batch_head, ntasks);
        for (int i = 0; i < threads-1; i++){
            cum_count[i] += offset;
        }
    }
    __syncthreads();
    if (tid > 0) {
        offset = cum_count[tid-1];
    }
    return offset;
}


extern "C" __global__ 
void screen_jk_tasks(ushort4 *shl_quartet_idx, int *batch_head, const int nbas, 
    const int * __restrict__ tile_ij_mapping, 
    const int * __restrict__ tile_kl_mapping, 
    const int ntiles_ij1, const int ntiles_kl1,
    const float * __restrict__ q_cond,
    const float * __restrict__ dm_cond, 
    const float cutoff, const float cutoff_fp64)
{
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    int ij = blockIdx.x * blockDim.x + tx;
    int kl = blockIdx.y * blockDim.y + ty;

    bool active = true;
    if (ij >= ntiles_ij1 || kl >= ntiles_kl1){
        ij = 0;
        kl = 0;
        active = false;
    }

    const int tile_ij = tile_ij_mapping[ij];
    const int tile_kl = tile_kl_mapping[kl];
    
    const int nbas_tiles = nbas / TILE;
    const int tile_i = tile_ij / nbas_tiles;
    const int tile_j = tile_ij % nbas_tiles;
    const int tile_k = tile_kl / nbas_tiles;
    const int tile_l = tile_kl % nbas_tiles;
    
    const int ish0 = tile_i * TILE;
    const int jsh0 = tile_j * TILE;
    const int ksh0 = tile_k * TILE;
    const int lsh0 = tile_l * TILE;
    const int ish1 = ish0 + TILE;
    const int jsh1 = jsh0 + TILE;
    const int ksh1 = ksh0 + TILE;
    const int lsh1 = lsh0 + TILE;

    constexpr int mask_size = (TILE*TILE*TILE*TILE) / 64;
    uint64_t mask_bits_fp32[mask_size] = {0};
    uint64_t mask_bits_fp64[mask_size] = {0};

    int count_fp32 = 0;
    int count_fp64 = 0;
    if (active){
        for (int i = 0; i < TILE; ++i){
            const int ish = ish0 + i;
            for (int j = 0; j < TILE; ++j){
                const int jsh = jsh0 + j;
                if (jsh >= ish+1 || jsh >= jsh1) continue;
                const int bas_ij = ish * nbas + jsh;
                const float q_ij = q_cond [bas_ij];
                const float d_ij = dm_cond[bas_ij];
                for (int k = 0; k < TILE; ++k){
                    const int ksh = ksh0 + k;
                    if (ksh >= ish+1 || ksh >= ksh1) continue;
                    const float d_ik = dm_cond[ish*nbas+ksh];
                    const float d_jk = dm_cond[jsh*nbas+ksh];
                    for (int l = 0; l < TILE; ++l){
                        const int lsh = lsh0 + l;
                        if (lsh >= ksh+1 || lsh >= lsh1) continue;
                        const int bas_kl = ksh * nbas + lsh;
                        if (bas_ij < bas_kl) continue;
                        const float q_ijkl = q_ij + q_cond[bas_kl];
                        float d_large = -36.8f;
                        if constexpr(do_k){
                            const float d_il = dm_cond[ish*nbas+lsh];
                            const float d_jl = dm_cond[jsh*nbas+lsh];
                            d_large = max(d_large, d_ik);
                            d_large = max(d_large, d_jk);
                            d_large = max(d_large, d_il);
                            d_large = max(d_large, d_jl);
                        }
                        if constexpr(do_j){
                            const float d_kl = dm_cond[bas_kl];
                            d_large = max(d_large, d_ij);
                            d_large = max(d_large, d_kl);
                        }
                        float dq = q_ijkl + d_large;
                        bool selected = (dq > cutoff) && (dq <= cutoff_fp64);
                        if (selected){
                            uint64_t idx = i*TILE*TILE*TILE + j*TILE*TILE + k*TILE + l;
                            uint64_t word = idx >> 6; // divide 64
                            uint64_t bit = idx & 63;
                            uint64_t bitmask = 1ull << bit;
                            mask_bits_fp32[word] |= bitmask;
                        }
                        count_fp32 += selected;
                        
                        selected = (dq > cutoff_fp64);
                        if (selected){
                            uint64_t idx = i*TILE*TILE*TILE + j*TILE*TILE + k*TILE + l;
                            uint64_t word = idx >> 6; // divide 64
                            uint64_t bit = idx & 63;
                            uint64_t bitmask = 1ull << bit;
                            mask_bits_fp64[word] |= bitmask;
                        }
                        count_fp64 += selected;
                    }
                }
            }
        }
    }
    int offset_fp32 = global_offset(batch_head+1, count_fp32);
    int offset_fp64 = global_offset(batch_head+2, -count_fp64) - 1;

    if (active){
#pragma unroll
        for (int i = 0; i < TILE; i++){
            for (int j = 0; j < TILE; j++){
                for (int k = 0; k < TILE; k++){
                    for (int l = 0; l < TILE; l++){
                        uint64_t idx = i*TILE*TILE*TILE + j*TILE*TILE + k*TILE + l;
                        uint64_t word = idx >> 6; // divide 64
                        uint64_t bit = idx & 63;
                        bool selected = (mask_bits_fp32[word] >> bit) & 1ull;
                        if (selected){
                            ushort4 sq;
                            sq.x = ish0 + i; 
                            sq.y = jsh0 + j; 
                            sq.z = ksh0 + k; 
                            sq.w = lsh0 + l;
                            shl_quartet_idx[offset_fp32] = sq;
                            ++offset_fp32;
                        }
                        selected = (mask_bits_fp64[word] >> bit) & 1ull;
                        if (selected){
                            ushort4 sq;
                            sq.x = ish0 + i; 
                            sq.y = jsh0 + j; 
                            sq.z = ksh0 + k; 
                            sq.w = lsh0 + l;
                            shl_quartet_idx[offset_fp64] = sq;
                            --offset_fp64;
                        }
                    }
                }
            }
        }
    }
}
